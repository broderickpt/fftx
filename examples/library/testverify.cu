#include "hip/hip_runtime.h"
#include <cmath> // Without this, abs returns zero!
#include <random>

#include "fftx_mddft_public.h"
#include "fftx_imddft_public.h"
#include "fftx_mdprdft_public.h"
#include "fftx_imdprdft_public.h"
// #include "fftx_rconv_public.h"

// Need this for AllSizes3 only
#include "fftx_imdprdft_decls.h"

#include "mddft.fftx.precompile.hpp"
#include "imddft.fftx.precompile.hpp"
#include "mdprdft.fftx.precompile.hpp"
#include "imdprdft.fftx.precompile.hpp"
// #include "rconv.fftx.precompile.hpp"

#include "fftx3utilities.h"

#include "device_macros.h"

enum VerbosityLevel { SHOW_CATEGORIES = 1, SHOW_SUBTESTS = 2, SHOW_ROUNDS = 3};
  
// using namespace fftx;

std::mt19937 generator;
// unifRealDist is uniform over the reals in (-1/2, 1/2).
std::uniform_real_distribution<double> unifRealDist;
// unifInt[d] is uniform over the integers in domain.lo[d] : domain.hi[d]
std::uniform_int_distribution<int> unifInt[3];

// Return random point in domain.
template<int DIM>
fftx::point_t<DIM> unifPoint()
{
  fftx::point_t<DIM> ret;
  for (int d = 0; d < DIM; d++)
    {
      ret[d] = unifInt[d](generator);
    }
  return ret;
}

// Return random real number.
double unifReal()
{
  return unifRealDist(generator);
}

// Return random complex number.
std::complex<double> unifComplex()
{
  return std::complex<double>(unifReal(), unifReal());
}

inline void getUnifScalar(double& a_scalar)
{
  a_scalar = unifReal();
}

inline void getUnifScalar(std::complex<double>& a_scalar)
{
  a_scalar = unifComplex();
}

template<typename T>
inline T unifScalar()
{
  T ret;
  getUnifScalar(ret);
  return ret;
}

template<typename T_IN, typename T_OUT>
void getUnifScalarPair(T_IN& a_scalarIn,
                       T_OUT& a_scalarOut);

void getUnifScalarPair(std::complex<double>& a_scalarIn,
                       std::complex<double>& a_scalarOut)
{
  a_scalarIn = unifComplex();
  a_scalarOut = a_scalarIn;
}

void getUnifScalarPair(double& a_scalarIn,
                       std::complex<double>& a_scalarOut)
{
  a_scalarIn = unifReal();
  a_scalarOut = std::complex<double>(a_scalarIn, 0.);
}

void getUnifScalarPair(std::complex<double>& a_scalarIn,
                       double& a_scalarOut)
                              
{
  a_scalarOut = unifReal();
  a_scalarIn = std::complex<double>(a_scalarOut, 0.);
}

// Fill a_arr with real numbers distributed uniformly in (-1/2, 1/2).
template<int DIM>
void unifRealArray(fftx::array_t<DIM, double>& a_arr)
{
  forall([](double(&v),
            const fftx::point_t<DIM>& p)
         {
           v = unifReal();
         }, a_arr);
}

// Fill a_arr with complex numbers with real and imaginary components distributed uniformly in (-1/2, 1/2).
template<int DIM>
void unifComplexArray(fftx::array_t<DIM, std::complex<double>>& a_arr)
{
  forall([](std::complex<double>(&v),
            const fftx::point_t<DIM>& p)
         {
           v = unifComplex();
         }, a_arr);
}

template<int DIM, typename T>
void unifArray(fftx::array_t<DIM, T>& a_arr);

template<int DIM>
void unifArray(fftx::array_t<DIM, double>& a_arr)
{
  unifRealArray(a_arr);
}

template<int DIM>
void unifArray(fftx::array_t<DIM, std::complex<double>>& a_arr)
{
  unifComplexArray(a_arr);
}

fftx::box_t<3> domainFromSize(const fftx::point_t<3>& a_size)
{
  fftx::box_t<3> bx(fftx::point_t<3>({{1, 1, 1}}),
                    fftx::point_t<3>({{a_size[0], a_size[1], a_size[2]}}));
  return bx;
}

template<int DIM>
size_t pointProduct(const fftx::point_t<DIM>& a_pt)
{
  size_t prod = 1;
  for (int d = 0; d < DIM; d++)
    {
      prod *= a_pt[d];
    }
  return prod;
}

// Set a_arr to a_scaling at point a_fixed, and 0 elsewhere.
template<int DIM, typename T>
void setUnitImpulse(fftx::array_t<DIM, T>& a_arr,
                    const fftx::point_t<DIM>& a_fixed,
                    T a_scaling = scalarVal<T>(1.) )
{
  forall([a_fixed, a_scaling](T(&v),
                              const fftx::point_t<DIM>& p)
         {
           if (p == a_fixed)
             {
               v = a_scaling;
             }
           else
             {
               v = scalarVal<T>(0.);
             }
         }, a_arr);
}

// Set a_arr to product of waves from impulse at a_fixed.
template<int DIM>
void setProductWaves(fftx::array_t<DIM, std::complex<double>>& a_arr,
                     const fftx::point_t<DIM>& a_extent,
                     const fftx::point_t<DIM>& a_fixed,
                     int a_sign)
{
  fftx::point_t<DIM> lo = a_arr.m_domain.lo;
  std::complex<double> omega[DIM];
  for (int d = 0; d < DIM; d++)
    {
      double th = (a_sign*2*(a_fixed[d] - lo[d])) * M_PI / (a_extent[d] * 1.);
      omega[d] = std::complex<double>(cos(th), sin(th));
    }
  forall([omega, lo](std::complex<double>(&v),
                     const fftx::point_t<DIM>& p)
         {
           v = std::complex<double>(1., 0.);
           for (int d = 0; d < DIM; d++)
             {
               v *= pow(omega[d], p[d] - lo[d]);
             }
         }, a_arr);
}

template<int DIM>
void setRotator(fftx::array_t<DIM, std::complex<double>>& a_arr,
                const fftx::box_t<DIM>& a_dom,
                int a_dim,
                int a_shift)
{
  fftx::point_t<DIM> lo = a_dom.lo;
  fftx::point_t<DIM> hi = a_dom.hi;
  fftx::point_t<DIM> fixed = lo;
  if (a_shift > 0)
    {
      fixed[a_dim] = lo[a_dim] + a_shift;
    }
  else if (a_shift < 0)
    {
      fixed[a_dim] = hi[a_dim] - (a_shift+1);
    }
  // std::cout << "setRotator in " << a_dim << " shift " << a_shift
  // << " waves " << fixed << " of " << a_arr.m_domain << std::endl;
  setProductWaves(a_arr, a_dom.extents(), fixed, -1);
}

template<int DIM, typename T_IN, typename T_OUT, class Transformer>
void TransformDevice(Transformer& a_tfm,
                     fftx::array_t<DIM, T_IN>& a_input, // make this const?
                     fftx::array_t<DIM, T_OUT>& a_output)

{
  auto inputDomain = a_input.m_domain;
  auto outputDomain = a_output.m_domain;

  auto input_size = inputDomain.size();
  auto output_size = outputDomain.size();

  auto input_bytes = input_size * sizeof(T_IN);
  auto output_bytes = output_size * sizeof(T_OUT);

  char* bufferPtr;
  DEVICE_MALLOC(&bufferPtr, input_bytes + output_bytes);
  T_IN* inputPtr = (T_IN*) bufferPtr;
  bufferPtr += input_bytes;
  T_OUT* outputPtr = (T_OUT*) bufferPtr;

  DEVICE_MEM_COPY(inputPtr, a_input.m_data.local(), input_bytes,
                  MEM_COPY_HOST_TO_DEVICE);

  fftx::array_t<DIM, T_IN> inputDevice(fftx::global_ptr<T_IN>
                                       (inputPtr, 0, 1), inputDomain);
  fftx::array_t<DIM, T_OUT> outputDevice(fftx::global_ptr<T_OUT>
                                         (outputPtr, 0, 1), outputDomain);

  a_tfm.transform(inputDevice, outputDevice);

  DEVICE_MEM_COPY(a_output.m_data.local(), outputPtr, output_bytes,
                  MEM_COPY_DEVICE_TO_HOST);
}

template<int DIM, typename T_IN, typename T_OUT, class Transformer>
double test1Transform(Transformer& a_tfm,
                      int a_rounds,
                      int a_verbosity)
{
  fftx::box_t<DIM> inputDomain = domainFromSize(a_tfm.inputSize());
  fftx::box_t<DIM> outputDomain = domainFromSize(a_tfm.outputSize());
  
  fftx::array_t<DIM, T_IN> inA(inputDomain);
  fftx::array_t<DIM, T_IN> inB(inputDomain);
  fftx::array_t<DIM, T_IN> LCin(inputDomain);

  fftx::array_t<DIM, T_OUT> outA(outputDomain);
  fftx::array_t<DIM, T_OUT> outB(outputDomain);
  fftx::array_t<DIM, T_OUT> LCout(outputDomain);
  fftx::array_t<DIM, T_OUT> outLCin(outputDomain);

  double errtest1 = 0.;
  for (int itn = 1; itn <= a_rounds; itn++)
    {
      T_IN alphaIn, betaIn;
      T_OUT alphaOut, betaOut;
      getUnifScalarPair(alphaIn, alphaOut);
      getUnifScalarPair(betaIn, betaOut);
      unifArray(inA);
      unifArray(inB);
      sumArrays(LCin, inA, inB, alphaIn, betaIn);

      TransformDevice(a_tfm, inA, outA);
      TransformDevice(a_tfm, inB, outB);
      sumArrays(LCout, outA, outB, alphaOut, betaOut);
      TransformDevice(a_tfm, LCin, outLCin);
      double err = absMaxDiffArray(outLCin, LCout);
      updateMax(errtest1, err);
      if (a_verbosity >= SHOW_ROUNDS)
        {
          printf("%dD linearity test round %d max error %11.5e\n", DIM, itn, err);
        }
    }
  if (a_verbosity >= SHOW_CATEGORIES)
    {
      printf("%dD Test 1 (linearity) in %d rounds: max error %11.5e\n", DIM, a_rounds, errtest1);
    }
  return errtest1;
}


template<int DIM, typename T_IN, typename T_OUT, class Transformer>
double test2impulse1(Transformer& a_tfm,
                     int a_verbosity)
{ // Unit impulse at low corner.
  fftx::box_t<DIM> inputDomain = domainFromSize(a_tfm.inputSize());
  fftx::box_t<DIM> outputDomain = domainFromSize(a_tfm.outputSize());

  fftx::array_t<DIM, T_IN> inImpulse(inputDomain);
  fftx::array_t<DIM, T_OUT> outImpulse(outputDomain);
  fftx::array_t<DIM, T_OUT> all1out(outputDomain);
  setUnitImpulse(inImpulse, inputDomain.lo);
  setConstant(all1out, scalarVal<T_OUT>(1.));
  TransformDevice(a_tfm, inImpulse, outImpulse);
  double errtest2impulse1 = absMaxDiffArray(outImpulse, all1out);
  if (a_verbosity >= SHOW_SUBTESTS)
    {
       printf("%dD unit impulse low corner test: max error %11.5e\n",
              DIM, errtest2impulse1);
    }
  return errtest2impulse1;
}

template<int DIM, typename T_IN, typename T_OUT, class Transformer>
double test2impulsePlus(Transformer& a_tfm,
                        int a_rounds,
                        int a_verbosity)
{ // Unit impulse at low corner.
  fftx::box_t<DIM> inputDomain = domainFromSize(a_tfm.inputSize());
  fftx::box_t<DIM> outputDomain = domainFromSize(a_tfm.outputSize());

  fftx::array_t<DIM, T_IN> inImpulse(inputDomain);
  fftx::array_t<DIM, T_OUT> outImpulse(outputDomain);
  fftx::array_t<DIM, T_OUT> all1out(outputDomain);
  setUnitImpulse(inImpulse, inputDomain.lo);
  setConstant(all1out, scalarVal<T_OUT>(1.));
  TransformDevice(a_tfm, inImpulse, outImpulse);

  fftx::array_t<DIM, T_IN> inRand(inputDomain);
  fftx::array_t<DIM, T_IN> inImpulseMinusRand(inputDomain);

  fftx::array_t<DIM, T_OUT> outRand(outputDomain);
  fftx::array_t<DIM, T_OUT> outImpulseMinusRand(outputDomain);
  fftx::array_t<DIM, T_OUT> mysum(outputDomain);
  
  // Check that for random arrays inRand,
  // fft(inRand) + fft(inImpulse - inRand) = fft(inImpulse) = all1out.
  double errtest2impulsePlus = 0.;
  for (int itn = 1; itn <= a_rounds; itn++)
    {
      unifArray(inRand);
      TransformDevice(a_tfm, inRand, outRand);
      diffArrays(inImpulseMinusRand, inImpulse, inRand);
      TransformDevice(a_tfm, inImpulseMinusRand, outImpulseMinusRand);
      sumArrays(mysum, outRand, outImpulseMinusRand);
      double err = absMaxDiffArray(mysum, all1out);
      updateMax(errtest2impulsePlus, err);
      if (a_verbosity >= SHOW_ROUNDS)
          {
            printf("%dD random + unit impulse low corner test round %d max error %11.5e\n", DIM, itn, err);
          }
    }

  if (a_verbosity >= SHOW_SUBTESTS)
    {
      printf("%dD unit impulse low corner test in %d rounds: max error %11.5e\n",
             DIM, a_rounds, errtest2impulsePlus);
    }
  return errtest2impulsePlus;
}

template<int DIM, typename T_IN, typename T_OUT, class Transformer>
double test2constant(Transformer& a_tfm,
                     int a_verbosity)
{ // Check that constant maps back to unit impulse at low corner.
  fftx::box_t<DIM> inputDomain = domainFromSize(a_tfm.inputSize());
  fftx::box_t<DIM> outputDomain = domainFromSize(a_tfm.outputSize());
  fftx::array_t<DIM, T_IN> all1in(inputDomain);
  setConstant(all1in, scalarVal<T_IN>(1.));

  fftx::array_t<DIM, T_OUT> magImpulse(outputDomain);
  size_t npts = pointProduct(a_tfm.size());
  T_OUT mag = scalarVal<T_OUT>(npts * 1.);
  setUnitImpulse(magImpulse, outputDomain.lo, mag);

  fftx::array_t<DIM, T_OUT> outImpulse(outputDomain);
  TransformDevice(a_tfm, all1in, outImpulse);

  double errtest2constant = absMaxDiffArray(outImpulse, magImpulse);
  if (a_verbosity >= SHOW_SUBTESTS)
    {
       printf("%dD constant test: max error %11.5e\n", DIM, errtest2constant);
    }
  return errtest2constant;
}

template<int DIM, typename T_IN, typename T_OUT, class Transformer>
double test2constantPlus(Transformer& a_tfm,
                         int a_rounds,
                         int a_verbosity)
{
  fftx::box_t<DIM> inputDomain = domainFromSize(a_tfm.inputSize());
  fftx::box_t<DIM> outputDomain = domainFromSize(a_tfm.outputSize());

  fftx::array_t<DIM, T_IN> all1in(inputDomain);
  setConstant(all1in, scalarVal<T_IN>(1.));

  fftx::array_t<DIM, T_OUT> magImpulse(outputDomain);
  size_t npts = pointProduct(a_tfm.size());
  T_OUT mag = scalarVal<T_OUT>(npts * 1.);
  setUnitImpulse(magImpulse, outputDomain.lo, mag);

  fftx::array_t<DIM, T_IN> inRand(inputDomain);
  fftx::array_t<DIM, T_IN> inConstantMinusRand(inputDomain);

  fftx::array_t<DIM, T_OUT> outRand(outputDomain);
  fftx::array_t<DIM, T_OUT> outConstantMinusRand(outputDomain);
  fftx::array_t<DIM, T_OUT> outSum(outputDomain);

  // Check that for random arrays inRand,
  // fft(inRand) + fft(all1 - inRand) = fft(all1) = magImpulse.
  double errtest2constantPlus = 0.;
  for (int itn = 1; itn <= a_rounds; itn++)
    {
      unifArray(inRand);
      TransformDevice(a_tfm, inRand, outRand);

      diffArrays(inConstantMinusRand, all1in, inRand);
      TransformDevice(a_tfm, inConstantMinusRand, outConstantMinusRand);

      sumArrays(outSum, outRand, outConstantMinusRand);
      
      double err = absMaxDiffArray(outSum, magImpulse);
      updateMax(errtest2constantPlus, err);
      if (a_verbosity >= SHOW_ROUNDS)
          {
            printf("%dD random + constant test round %d max error %11.5e\n",
                   DIM, itn, err);
          }
    }

  if (a_verbosity >= SHOW_SUBTESTS)
    {
      printf("%dD random + constant test in %d rounds: max error %11.5e\n",
             DIM, a_rounds, errtest2constantPlus);
  
    }
  return errtest2constantPlus;
}

template<int DIM, typename T_IN, typename T_OUT, class Transformer>
double test2impulseRandom(Transformer& a_tfm,
                          int a_sign,
                          int a_rounds,
                          int a_verbosity)
{
  // Do nothing if T_OUT is real. Run this test only if T_OUT is complex.
  return 0.;
}


template<int DIM, typename T_IN, class Transformer>
double test2impulseRandom(Transformer& a_tfm,
                          int a_sign,
                          int a_rounds,
                          int a_verbosity)
{
  // Check unit impulse at random position.
  fftx::box_t<DIM> inputDomain = domainFromSize(a_tfm.inputSize());
  fftx::box_t<DIM> outputDomain = domainFromSize(a_tfm.outputSize());

  fftx::array_t<DIM, T_IN> inImpulse(inputDomain);
  fftx::array_t<DIM, std::complex<double>> outImpulse(outputDomain); // FIXME
  fftx::array_t<DIM, std::complex<double>> outCheck(outputDomain); // FIXME
  double errtest2impulseRandom = 0.;
  fftx::point_t<DIM> fullExtents = a_tfm.size();
  for (int itn = 1; itn <= a_rounds; itn++)
    {
      fftx::point_t<DIM> rpoint = unifPoint<DIM>();
      setUnitImpulse(inImpulse, rpoint);
      TransformDevice(a_tfm, inImpulse, outImpulse);
      // Recall inputDomain is whole domain,
      // but outputDomain may be truncated;
      // waves defined on outputDomain,
      // but based on the full inputDomain extents.
      setProductWaves(outCheck, fullExtents, rpoint, a_sign);
      double err = absMaxDiffArray(outImpulse, outCheck);
      updateMax(errtest2impulseRandom, err);
      if (a_verbosity >= SHOW_ROUNDS)
        {
          printf("%dD random impulse test round %d max error %11.5e\n", DIM, itn, err);
        }
    }
  return errtest2impulseRandom;
}


template<int DIM, typename T_IN, typename T_OUT, class Transformer>
double test2Transform(Transformer& a_tfm,
                      int a_sign,
                      int a_rounds,
                      int a_verbosity)
{
  double errtest2 = 0.;

  updateMax(errtest2,
            test2impulse1<DIM, T_IN, T_OUT, Transformer>
            (a_tfm, a_verbosity));

  updateMax(errtest2,
            test2impulsePlus<DIM, T_IN, T_OUT, Transformer>
            (a_tfm, a_rounds, a_verbosity));

  updateMax(errtest2,
            test2constant<DIM, T_IN, T_OUT, Transformer>
            (a_tfm, a_verbosity));
  
  updateMax(errtest2,
            test2constantPlus<DIM, T_IN, T_OUT, Transformer>
            (a_tfm, a_rounds, a_verbosity));
  
  updateMax(errtest2,
            test2impulseRandom<DIM, T_IN, T_OUT, Transformer>
            (a_tfm, a_sign, a_rounds, a_verbosity));
  
  if (a_verbosity >= SHOW_CATEGORIES)
    {
      printf("%dD Test 2 (impulses) in %d rounds: max error %11.5e\n", DIM, a_rounds, errtest2);
    }
  return errtest2;
}


template<int DIM, typename T_IN, typename T_OUT, class Transformer>
double test3time(Transformer& a_tfm,
                 int a_sign,
                 int a_rounds,
                 int a_verbosity)
{
  // Do nothing if T_OUT is real. Run this test only if T_OUT is complex.
  return 0.;
}

template<int DIM, typename T_IN, class Transformer>
double test3time(Transformer& a_tfm,
                 int a_sign,
                 int a_rounds,
                 int a_verbosity)
{
  fftx::box_t<DIM> inputDomain = domainFromSize(a_tfm.inputSize());
  fftx::box_t<DIM> outputDomain = domainFromSize(a_tfm.outputSize());

  fftx::array_t<DIM, T_IN> inRand(inputDomain);
  fftx::array_t<DIM, T_IN> inRandRot(inputDomain);
  fftx::array_t<DIM, std::complex<double>> outRand(outputDomain);
  fftx::array_t<DIM, std::complex<double>> outRandRot(outputDomain);
  fftx::array_t<DIM, std::complex<double>> rotator(outputDomain);
  fftx::array_t<DIM, std::complex<double>> outRandRotMult(outputDomain);
  double errtest3timeDim[DIM];
  double errtest3time = 0.;
  for (int d = 0; d < DIM; d++)
    {
      errtest3timeDim[d] = 0.;
      setRotator(rotator, inputDomain, d, -a_sign); // +1 for MDDFT, -1 for IMDDFT, -1 for PRDFT
      for (int itn = 1; itn <= a_rounds; itn++)
        {
          unifArray(inRand);
          
          // time-shift test in dimension d
          rotate(inRandRot, inRand, d, 1); // +1 for MDDFT, +1 for IMDDFT, +1 for PRDFT
          TransformDevice(a_tfm, inRand, outRand);
          TransformDevice(a_tfm, inRandRot, outRandRot);
          productArrays(outRandRotMult, outRandRot, rotator);
          double err = absMaxDiffArray(outRandRotMult, outRand);
          updateMax(errtest3timeDim[d], err);
          updateMax(errtest3time, errtest3timeDim[d]);
          if (a_verbosity >= SHOW_ROUNDS)
            {
              printf("%dD dim %d time-shift test %d max error %11.5e\n",
                     DIM, d, itn, err);
            }
        }
      if (a_verbosity >= SHOW_SUBTESTS)
        {
          printf("%dD dim %d time-shift test in %d rounds: max error %11.5e\n",
                 DIM, d, a_rounds, errtest3timeDim[d]);
        }
    }
  return errtest3time;
}

template<int DIM, typename T_IN, typename T_OUT, class Transformer>
double test3frequency(Transformer& a_tfm,
                      int a_sign,
                      int a_rounds,
                      int a_verbosity)
{
  // Do nothing if T_IN is real. Run this test only if T_IN is complex.
  return 0.;
}

template<int DIM, typename T_OUT, class Transformer>
double test3frequency(Transformer& a_tfm,
                      int a_sign,
                      int a_rounds,
                      int a_verbosity)
{
  fftx::box_t<DIM> inputDomain = domainFromSize(a_tfm.inputSize());
  fftx::box_t<DIM> outputDomain = domainFromSize(a_tfm.outputSize());

  fftx::array_t<DIM, std::complex<double>> inRand(inputDomain);
  fftx::array_t<DIM, std::complex<double>> inRandMult(inputDomain);
  fftx::array_t<DIM, T_OUT> outRand(outputDomain);
  fftx::array_t<DIM, T_OUT> outRandMult(outputDomain);
  fftx::array_t<DIM, std::complex<double>> rotatorUp(inputDomain);
  fftx::array_t<DIM, T_OUT> outRandMultRot(outputDomain);
  double errtest3frequencyDim[DIM];
  double errtest3frequency = 0.;
  for (int d = 0; d < DIM; d++)
    {
      // frequency-shift test in dimension d
      errtest3frequencyDim[d] = 0.;
      // Recall outputDomain is whole domain,
      // but inputDomain may be truncated;
      // rotatorUp is defined on inputDomain,
      // but based on full outputDomain.
      setRotator(rotatorUp, outputDomain, d, 1);
      for (int itn = 1; itn <= a_rounds; itn++)
        {
          unifComplexArray(inRand);

          productArrays(inRandMult, inRand, rotatorUp);
          TransformDevice(a_tfm, inRand, outRand);
          TransformDevice(a_tfm, inRandMult, outRandMult);
          rotate(outRandMultRot, outRandMult, d, a_sign);
          double err = absMaxDiffArray(outRandMultRot, outRand);
          updateMax(errtest3frequencyDim[d], err);
          updateMax(errtest3frequency, errtest3frequencyDim[d]);
          if (a_verbosity >= SHOW_ROUNDS)
            {
              printf("%dD dim %d frequency-shift test %d max error %11.5e\n", DIM, d, itn, err);
            }
        }
      if (a_verbosity >= SHOW_SUBTESTS)
        {
          printf("%dD dim %d frequency-shift test in %d rounds: max error %11.5e\n", DIM, d, a_rounds, errtest3frequencyDim[d]);
        }
    }
  return errtest3frequency;
}

template<int DIM, typename T_IN, typename T_OUT, class Transformer>
double test3Transform(Transformer& a_tfm,
                      int a_sign,
                      int a_rounds,
                      int a_verbosity)
{
  double errtest3 = 0.;

  updateMax(errtest3,
            test3time<DIM, T_IN, T_OUT, Transformer>
            (a_tfm, a_sign, a_rounds, a_verbosity));
  
  updateMax(errtest3,
            test3frequency<DIM, T_IN, T_OUT, Transformer>
            (a_tfm, a_sign, a_rounds, a_verbosity));
  
  if (a_verbosity >= SHOW_CATEGORIES)
    {
      printf("%dD Test 3 (shifts) in %d rounds: max error %11.5e\n",
             DIM, a_rounds, errtest3);
    }
  return errtest3;
}


template<int DIM, typename T_IN, typename T_OUT, class Transformer>
void verifyTransform(Transformer& a_tfm,
                     int a_sign,
                     int a_rounds,
                     int a_verbosity)
{
  double err = 0.;

  updateMax(err,
            test1Transform<DIM, T_IN, T_OUT, Transformer>
            (a_tfm, a_rounds, a_verbosity));

  updateMax(err,
            test2Transform<DIM, T_IN, T_OUT, Transformer>
            (a_tfm, a_sign, a_rounds, a_verbosity));

  updateMax(err,
            test3Transform<DIM, T_IN, T_OUT, Transformer>
            (a_tfm, a_sign, a_rounds, a_verbosity));

  printf("%dD test on %s in %d rounds max error %11.5e\n",
         DIM, a_tfm.name().c_str(), a_rounds, err);
}


int main(int argc, char* argv[])
{
  // { SHOW_CATEGORIES = 1, SHOW_SUBTESTS = 2, SHOW_ROUNDS = 3};
  printf("Usage:  %s [verbosity=0] [rounds=20]\n", argv[0]);
  printf("verbosity 0 for summary, 1 for categories, 2 for subtests, 3 for rounds\n");
  int verbosity = 0;
  int rounds = 20;
  if (argc > 1)
    {
      verbosity = atoi(argv[1]);
      if (argc > 2)
        {
          rounds = atoi(argv[2]);
        }
    }
  printf("Running with verbosity %d, random %d rounds\n", verbosity, rounds);

  /*
    Set up random number generator.
  */
  std::random_device rd;
  generator = std::mt19937(rd());
  unifRealDist = std::uniform_real_distribution<double>(-0.5, 0.5);

    // last entry is { 0, 0, 0 }
  int numentries = sizeof ( AllSizes3 ) / sizeof ( fftx::point_t<3> ) - 1;
  
  for ( int ind = 0; ind < numentries; ind++ )
    {
      fftx::point_t<3> sz = AllSizes3[ind];

      for (int d = 0; d < 3; d++)
        {
          unifInt[d] = std::uniform_int_distribution<int>(1, sz[d]);
        }

      {
        fftx::mddft<3> tfm(sz);
        verifyTransform
          <3, std::complex<double>, std::complex<double>, fftx::mddft<3>>
          (tfm, -1, rounds, verbosity);
       }

      {
        fftx::imddft<3> tfm(sz);
        verifyTransform
          <3, std::complex<double>, std::complex<double>, fftx::imddft<3>>
          (tfm, 1, rounds, verbosity);
       }

      {
        fftx::mdprdft<3> tfm(sz);
        verifyTransform
          <3, double, std::complex<double>, fftx::mdprdft<3>>
          (tfm, -1, rounds, verbosity);
      }

      {
        fftx::imdprdft<3> tfm(sz);
        verifyTransform
          <3, std::complex<double>, double, fftx::imdprdft<3>>
          (tfm, 1, rounds, verbosity);
      }
    }

  printf("%s: All done, exiting\n", argv[0]);
  return 0;
}
